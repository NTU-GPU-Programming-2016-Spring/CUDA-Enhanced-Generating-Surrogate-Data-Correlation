#include "hip/hip_runtime.h"
/* For Brian */
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "Timer.h"

#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hiprand.h>

#define SIGSIZE 440
#define SIGDIM 10000
#define NBLK 256
#define TIMESLOT 439
#define pi 3.14159

// 'aaft' is output.
void amplitudeAdjustedFourierTransform(double *aaft, const double *data, const int row, const int maxColumns) {
	
	return;
}

__device__ float angle_trans(const hipComplex& z){
	return atan2(hipCimagf(z), hipCrealf(z));
}

__global__ void fft_polar_angle(hipfftComplex *data, float *angle, float *mag, int data_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	//abs of fft
	mag[idx] = hipCabsf(data[idx]);
	//angle of fft
	angle[idx] = angle_trans(data[idx]);
	return;
}

// do p(2:N)=[p1 -flipud(p1)];
__global__ void odd_surr_trans(float *angle, float *ran, int data_size, int sig_size, int half_sig_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	

	int data_col = idx/sig_size;
	int data_idx = idx%sig_size;
	// p(1) is not necessary for changing
	if(data_idx ==0){
		return;
	}
	
	int half_idx;
	//p(2: 2+half-1)
	if(data_idx <= half_sig_size){
		half_idx = (data_idx-1) + data_col*half_sig_size;
		angle[idx] = 2*pi*ran[half_idx];
			
	// -flipup(p1)	
	}else{
		int diff = data_idx - half_sig_size;
		int reverse_data_idx = half_sig_size- diff;
		half_idx = reverse_data_idx + data_col*half_sig_size;
		angle[idx] = -2*pi*ran[half_idx];
	}

	return;

}

__global__ void even_surr_trans(float *angle, float *mag, float *ran, int data_size, int sig_size, int half_sig_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	
	int data_col = idx/sig_size;
	int data_idx = idx%sig_size;
	// angle part
	// p(2:N)=[p1' p(h+1) -flipud(p1)'];
	int half_idx;
	// 0 nothing
	// 1->half_sig_size-1
	if(data_idx == 0 || data_idx == half_sig_size){
		angle[idx] = angle[idx];
	}
	else if(1<= data_idx < half_sig_size){
		half_idx = (data_idx-1) + data_col*half_sig_size;
		angle[idx] = 2*pi*ran[half_idx];
	}
	// half_sig_size
	// half_sig_size+1->data_size-1
	if(data_idx > half_sig_size){
		int diff = data_idx - half_sig_size+1;
		int reverse_data_idx = half_sig_size- diff;
		
		half_idx = reverse_data_idx + data_col*half_sig_size;
		angle[idx] = -2*pi*ran[half_idx];

		// magnitude part
		// m=[flipud(m(2:h))];
		diff = data_idx - (half_sig_size);
		reverse_data_idx = (half_sig_size) - diff;
		int mag_idx = reverse_data_idx + data_col*sig_size;
		mag[idx] = mag[mag_idx];
	}

	return;

}
// s(:,j)=m.*exp(i*p);
__global__ void i_mul_trans(hipfftComplex *result, const float *mag, const float *angle, int data_size){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx >= data_size){
		return;
	}
	float mag_val = mag[idx];
	float angle_val = angle[idx];
	result[idx].x = mag_val*cosf(angle_val);
	result[idx].y = mag_val*sinf(angle_val);

	return;
}

__global__ void get_real_trans(float *result, const hipfftComplex *data_list, const int data_size, const int sig_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	if (idx >= data_size){
		return;
	}
	result[idx] = data_list[idx].x/sig_size;
	return;
}

__global__ void real2cufft_trans(hipfftComplex *result, const float *input, const int data_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	if (idx >= data_size){
		return;
	}
	result[idx].x = input[idx];
	result[idx].y = 0;
	return;
}

void phaseran(float *result, const int data_num, const int time_size){
	int data_size = data_num*time_size;
	int mem_size = sizeof(hipfftComplex)*data_size;
	
	hipfftComplex *d_signal;
	checkCudaErrors(hipMalloc((void **) &d_signal, mem_size));
	float *d_input;
	checkCudaErrors(hipMalloc(&d_input, sizeof(float)*data_size));
	checkCudaErrors(hipMemcpy(d_input, result, sizeof(float)*data_size, hipMemcpyHostToDevice));
	
	real2cufft_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_signal, d_input, data_size);
	
	//cufft
	hipfftHandle plan_r, plan;
	
	if (hipfftPlan1d(&plan_r, time_size, HIPFFT_R2C, data_num) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}
	if (hipfftPlan1d(&plan, time_size, HIPFFT_C2C, data_num) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}

	//forward transform
	// printf("---Transform fft--- \n");
	hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);
	checkCudaErrors(hipFree(d_input));
	
	//do angle implement in matlab
	float *d_angle, *d_mag;
	checkCudaErrors(hipMalloc(&d_angle, sizeof(float)*data_size));
	checkCudaErrors(hipMalloc(&d_mag, sizeof(float)*data_size));

	fft_polar_angle<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_signal, d_angle, d_mag, data_size);

	checkCudaErrors(hipFree(d_signal));

	// start parallel surrogate
	int half_col_size = time_size/2;
	int half_size = half_col_size*data_num;
	float *d_ran_series;
	
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, rand()%10000);
	
	if(time_size%2==0){
		//assign half minus 1
		int half_minus_one_size = (half_col_size-1)*data_num;
		checkCudaErrors(hipMalloc(&d_ran_series, sizeof(float)*half_minus_one_size));
		hiprandGenerateUniform(gen, d_ran_series, half_minus_one_size);

		even_surr_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_angle, d_mag, d_ran_series, data_size, time_size, half_col_size);
	
	}else{		
		
		//assign half 
		checkCudaErrors(hipMalloc(&d_ran_series, sizeof(float)*half_size));
		//random generator
		
		hiprandGenerateUniform(gen, d_ran_series, half_size);
		
		// do column vector trans p(2:N)=[p1 -flipud(p1)];
		odd_surr_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_angle, d_ran_series, data_size, time_size, half_col_size);
	}
	checkCudaErrors(hipFree(d_ran_series));
	
	// multiply with m.*exp(i*p) = m*cos(p) + m*i*sin(p)
	hipfftComplex *d_i_mul;
	checkCudaErrors(hipMalloc((void **) &d_i_mul, mem_size));
	
	i_mul_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_i_mul, d_mag, d_angle, data_size);

	// backward transform
	// printf("---Inverse fft transform --- \n");
	hipfftExecC2C(plan, d_i_mul, d_i_mul, 
							   HIPFFT_BACKWARD);
	float *d_result;
	checkCudaErrors(hipMalloc(&d_result, sizeof(float)*data_size));

	get_real_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_result, d_i_mul, data_size, time_size);
	
	checkCudaErrors(hipMemcpy(result, d_result, sizeof(float)*data_size, hipMemcpyDeviceToHost));
	
	hipfftDestroy(plan);
	
	checkCudaErrors(hipFree(d_angle));
	checkCudaErrors(hipFree(d_mag));
	checkCudaErrors(hipFree(d_result));
	checkCudaErrors(hipFree(d_i_mul));
	hipDeviceReset();
	return;
}

int main(int argc, char **argv)
{	
	float *result = (float *)malloc(sizeof(float)*SIGSIZE*SIGDIM);	
	Timer phaseran_timer;
	phaseran_timer.Start();
	for(int i = 0; i <1 ; i++){

		for(int i = 0; i<SIGSIZE*SIGDIM;i++){
			result[i] = i;
		}
		
		phaseran(result, SIGDIM, SIGSIZE);
		
	}
	phaseran_timer.Pause();
	printf_timer(phaseran_timer);
	free(result);
	return 0;
}	