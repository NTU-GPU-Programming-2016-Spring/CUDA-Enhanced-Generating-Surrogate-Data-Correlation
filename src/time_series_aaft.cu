#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// #include "Timer.h"

#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hiprand.h>

#define SIGSIZE 439
#define SIGDIM 10000
#define NBLK 256
#define TIMESLOT 439
#define pi 3.14159

__device__ float angle_trans(const hipComplex& z){
	return atan2(hipCimagf(z), hipCrealf(z));
}

__global__ void fft_polar_angle(hipfftComplex *data, float *angle, float *mag, int data_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	//abs of fft
	mag[idx] = hipCabsf(data[idx]);
	//angle of fft
	angle[idx] = angle_trans(data[idx]);
	return;
}

// do p(2:N)=[p1 -flipud(p1)];
__global__ void odd_surr_trans(float *angle, float *ran, int data_size, int sig_size, int half_sig_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	
	int data_col = idx/sig_size;
	int data_idx = idx%sig_size;
	// p(1) is not necessary for changing
	if(data_idx ==0){
		return;
	}
	
	int half_idx;
	//p(2: 2+half-1)
	if(data_idx <= half_sig_size){
		half_idx = (data_idx-1) + data_col*half_sig_size;
		angle[idx] = 2*pi*ran[half_idx];
			
	// -flipup(p1)	
	}else{
		int diff = data_idx - half_sig_size;
		int reverse_data_idx = half_sig_size- diff;
		half_idx = reverse_data_idx + data_col*half_sig_size;
		angle[idx] = -2*pi*ran[half_idx];
	}

	return;
}

__global__ void even_surr_trans(float *angle, float *mag, float *ran, int data_size, int sig_size, int half_sig_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= data_size){
		return;
	}
	
	int data_col = idx/sig_size;
	int data_idx = idx%sig_size;
	// angle part
	// p(2:N)=[p1' p(h+1) -flipud(p1)'];
	int half_idx;
	// 0 nothing
	// 1->half_sig_size-1
	if(data_idx == 0 || data_idx == half_sig_size){
		angle[idx] = angle[idx];
	}
	else if(1<= data_idx < half_sig_size){
		half_idx = (data_idx-1) + data_col*half_sig_size;
		angle[idx] = 2*pi*ran[half_idx];
	}
	// half_sig_size
	// half_sig_size+1->data_size-1
	if(data_idx > half_sig_size){
		int diff = data_idx - half_sig_size+1;
		int reverse_data_idx = half_sig_size- diff;
		
		half_idx = reverse_data_idx + data_col*half_sig_size;
		angle[idx] = -2*pi*ran[half_idx];

		// magnitude part
		// m=[flipud(m(2:h))];
		diff = data_idx - (half_sig_size);
		reverse_data_idx = (half_sig_size) - diff;
		int mag_idx = reverse_data_idx + data_col*sig_size;
		mag[idx] = mag[mag_idx];
	}

	return;

}
// s(:,j)=m.*exp(i*p);
__global__ void i_mul_trans(hipfftComplex *result, const float *mag, const float *angle, int data_size){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx >= data_size){
		return;
	}
	float mag_val = mag[idx];
	float angle_val = angle[idx];
	result[idx].x = mag_val*cosf(angle_val);
	result[idx].y = mag_val*sinf(angle_val);

	return;
}

__global__ void get_real_trans(float *result, const hipfftComplex *data_list, const int data_size, const int sig_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	if (idx >= data_size){
		return;
	}
	result[idx] = data_list[idx].x/sig_size;
	return;
}

__global__ void real2cufft_trans(hipfftComplex *result, const float *input, const int data_size){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	if (idx >= data_size){
		return;
	}
	result[idx].x = input[idx];
	result[idx].y = 0;
	return;
}

void phaseran(float *result, const int data_num, const int time_size){
	int data_size = data_num*time_size;
	int mem_size = sizeof(hipfftComplex)*data_size;
	
	hipfftComplex *d_signal;
	checkCudaErrors(hipMalloc((void **) &d_signal, mem_size));
	
	real2cufft_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_signal, result, data_size);
	hipDeviceSynchronize();

	//cufft
	hipfftHandle plan;
	
	if (hipfftPlan1d(&plan, time_size, HIPFFT_C2C, data_num) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
	}

	//forward transform
	// printf("---Transform fft--- \n");
	hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);
	
	//do angle implement in matlab
	float *d_angle, *d_mag;
	checkCudaErrors(hipMalloc(&d_angle, sizeof(float)*data_size));
	checkCudaErrors(hipMalloc(&d_mag, sizeof(float)*data_size));

	fft_polar_angle<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_signal, d_angle, d_mag, data_size);
	hipDeviceSynchronize();

	checkCudaErrors(hipFree(d_signal));

	// start parallel surrogate
	int half_col_size = time_size/2;
	int half_size = half_col_size*data_num;
	float *d_ran_series;
	
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, rand()%10000);
	
	if(time_size%2==0){
		//assign half minus 1
		int half_minus_one_size = (half_col_size-1)*data_num;
		checkCudaErrors(hipMalloc(&d_ran_series, sizeof(float)*half_minus_one_size));
		hiprandGenerateUniform(gen, d_ran_series, half_minus_one_size);

		even_surr_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_angle, d_mag, d_ran_series, data_size, time_size, half_col_size);
		
	}else{		
		
		//assign half 
		checkCudaErrors(hipMalloc(&d_ran_series, sizeof(float)*half_size));
		//random generator
		
		hiprandGenerateUniform(gen, d_ran_series, half_size);
		
		// do column vector trans p(2:N)=[p1 -flipud(p1)];
		odd_surr_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_angle, d_ran_series, data_size, time_size, half_col_size);
	}
	hipDeviceSynchronize();

	checkCudaErrors(hipFree(d_ran_series));
	hiprandDestroyGenerator(gen);
	
	// multiply with m.*exp(i*p) = m*cos(p) + m*i*sin(p)
	hipfftComplex *d_i_mul;
	checkCudaErrors(hipMalloc((void **) &d_i_mul, mem_size));
	
	i_mul_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(d_i_mul, d_mag, d_angle, data_size);
	hipDeviceSynchronize();

	// backward transform
	// printf("---Inverse fft transform --- \n");
	hipfftExecC2C(plan, d_i_mul, d_i_mul, 
							   HIPFFT_BACKWARD);
	get_real_trans<<<(data_size+NBLK-1)/NBLK, NBLK>>>(result, d_i_mul, data_size, time_size);
	hipDeviceSynchronize();
	
	hipfftDestroy(plan);
	
	checkCudaErrors(hipFree(d_angle));
	checkCudaErrors(hipFree(d_mag));
	checkCudaErrors(hipFree(d_i_mul));
	return;
}
// for thrust::generate's generator
int generator_mod_pivot = 0;
int generator_time_points = 0;

int mod_fuc(){
	return (generator_mod_pivot++)/generator_time_points;
}

template<class DataType>
void sortData(DataType *data, const int viewers, const int randomNum, const int timePoints){

	int total_size = viewers*randomNum*timePoints;
	// sort by key implement
	// vectorized sort
	thrust::host_vector<int> h_rank(total_size);
	
	// initialize # of randomNum pivot
	generator_mod_pivot = 0;
	thrust::generate(h_rank.begin(), h_rank.end(), mod_fuc);
  	thrust::device_vector<int> d_rank = h_rank;
  	thrust::device_ptr<DataType> d_result = thrust::device_pointer_cast(data);
  	// pointer to vector
  	thrust::device_vector<DataType> d_v_result(d_result, d_result+total_size);
  	  
  	thrust::stable_sort_by_key(d_v_result.begin(), d_v_result.end(), d_rank.begin());
  	thrust::stable_sort_by_key(d_rank.begin(), d_rank.end(), d_v_result.begin());
  	
  	hipDeviceSynchronize();

  	DataType *raw_ptr = thrust::raw_pointer_cast(d_v_result.data());
  	checkCudaErrors(hipMemcpy(data, raw_ptr, total_size*sizeof(DataType), hipMemcpyDeviceToDevice));
	
	return;
}
// get sort self-defined functions
struct sort_int2{
	__host__ __device__ bool operator()(const int2 &lhs, const int2 &rhs) const{
		return (lhs.x < rhs.x);  
	}
};

struct sort_2d_rank{
	__host__ __device__ bool operator()(const int2 &lhs, const int2 &rhs) const{
		return (lhs.x < rhs.x) || ((lhs.x == rhs.x) && (lhs.y < rhs.y));
	}
};
struct trans_1d{
	__host__ __device__ int operator()(const int2 &x) const{
		return x.y;
	}
};

struct trans_2d{
	int timePoints;
	trans_2d(int _timePoints) : timePoints(_timePoints){}

	__host__ __device__ int2 operator()(const int &idx) const{
		int rand_pivot = idx/timePoints;
		int init_posi = idx%timePoints+1;
		int2 temp = make_int2(rand_pivot, init_posi);

		return temp;
	}
};

struct rank_2d_trans{
	int timePoints;
	rank_2d_trans(int _timePoints) : timePoints(_timePoints){}

	__host__ __device__ int2 operator()(const int &ran_idx, const int &rank_idx) const{
		int rand_pivot = ran_idx/timePoints;
		int2 temp = make_int2(rand_pivot, rank_idx);

		return temp;
	}
};

// get sort rank in *rank
template<class DataType>
void getSortRank(int *rank, DataType *data, const int viewers, const int randomNum, const int timePoints){
	int total_size = viewers*randomNum*timePoints;
	
	// do rank
	thrust::device_vector<int2> d_rank(total_size);
	
	generator_mod_pivot = 0;
	// generate key
	thrust::device_vector<int> d_prep(total_size);
	thrust::sequence(d_prep.begin(), d_prep.end());
	thrust::transform(d_prep.begin(), d_prep.end(), d_rank.begin(), trans_2d(timePoints));
	
	// start get rank
	thrust::device_ptr<DataType> d_result = thrust::device_pointer_cast(data);
	thrust::device_vector<DataType> d_v_result(d_result, d_result+total_size);

	thrust::stable_sort_by_key(d_v_result.begin(), d_v_result.end(), d_rank.begin());
	thrust::stable_sort_by_key(d_rank.begin(), d_rank.end(), d_v_result.begin(), sort_int2());	

  	// do transform and copy back
  	thrust::device_vector<int> d_rank_ans(total_size);
  	thrust::transform(d_rank.begin(), d_rank.end(), d_rank_ans.begin(), trans_1d());
	checkCudaErrors(hipMemcpy(rank, d_rank_ans.data().get(), sizeof(int)*total_size, hipMemcpyDeviceToDevice));

  	return;
}

// sort data by given ranks
template<class DataType>
void sortByRank(DataType *data, int *rank, const int viewers, const int randomNum, const int timePoints){
	int total_size = viewers*randomNum*timePoints;
	thrust::device_vector<int> d_random_idx(total_size);
	thrust::sequence(d_random_idx.begin(), d_random_idx.end());
	
	thrust::device_ptr<int> rank_ptr(rank);
	thrust::device_vector<int> rank_vec(rank_ptr, rank_ptr+total_size);

	// construct 2d int2 (ran_idx, rank)
	thrust::device_vector<int2> int2_rank_vec(total_size);
	thrust::transform(d_random_idx.begin(), d_random_idx.end(), rank_vec.begin(), int2_rank_vec.begin(), rank_2d_trans(timePoints));

	// data to thrust vector
	thrust::device_ptr<DataType> data_ptr = thrust::device_pointer_cast(data);;
	thrust::device_vector<DataType> d_data_vec(data_ptr, data_ptr+total_size);

	thrust::stable_sort_by_key(int2_rank_vec.begin(), int2_rank_vec.end(), d_data_vec.begin(), sort_2d_rank());

	// test for correctness
	// thrust::host_vector<DataType> h_data_vec=d_data_vec;
	// thrust::device_ptr<DataType> odata_ptr = thrust::device_pointer_cast(data);;
	// thrust::device_vector<DataType> od_data_vec(odata_ptr, odata_ptr+total_size);
	// thrust::host_vector<DataType> oh_data_vec = od_data_vec;
	// thrust::host_vector<int> h_rank = rank_vec;

	// for(int i = 0 ; i< 2*timePoints; i++){
	// 	if(i%timePoints==0){
	// 		printf("-------\n");
	// 	}
	// 	printf("ori: %1f rank: %d sort: %1f \n", oh_data_vec[i], h_rank[i], h_data_vec[i]);
	// }

	checkCudaErrors(hipMemcpy(data, d_data_vec.data().get(), sizeof(DataType)*total_size, hipMemcpyDeviceToDevice));

	return;
}

// aaft : cudaPointer return value
// data : cudaPointer input data subjects (with several viewers)
// viewers : # of viewers in data
// randomNum : # of random series
// timePoints :  # of time slots
void amplitudeAdjustedFourierTransform(double *d_aaft, double *d_data, const int viewers, const int randomNum, const int timePoints) {
	// generate normal random variables
	generator_time_points = timePoints;
	int total_size = viewers*randomNum*timePoints;
	float *d_normal;
	checkCudaErrors(hipMalloc(&d_normal, sizeof(float)*total_size));
	// copy original d_data to d_aaft
	checkCudaErrors(hipMemcpy(d_aaft, d_data, sizeof(double)*total_size, hipMemcpyDeviceToDevice));

	// generator 
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, rand()%10000);
	hiprandGenerateNormal(gen, d_normal, total_size, 0, 1);
	hiprandDestroyGenerator(gen);
	
	// sort d_normal -> normal=sort(randn(size(y)));
	sortData(d_normal, viewers, randomNum, timePoints);
	
	int *d_rank;
	checkCudaErrors(hipMalloc(&d_rank, sizeof(int)*total_size));

	// [y,T]=sort(y);
	getSortRank(d_rank, d_data, viewers, randomNum, timePoints);
	
	// [T,r]=sort(T);
	getSortRank(d_rank, d_rank, viewers, randomNum, timePoints);
	
	// normal=phaseran(normal(r));
	sortByRank(d_normal, d_rank, viewers, randomNum, timePoints);
	phaseran(d_normal, viewers*randomNum, timePoints);

	// [normal,T]=sort(normal);
	getSortRank(d_rank, d_normal, viewers, randomNum, timePoints);
	
	// [T,r]=sort(T);
	getSortRank(d_rank, d_rank, viewers, randomNum, timePoints);

	// s=y(r);
	sortByRank(d_aaft, d_rank, viewers, randomNum, timePoints);
	
	checkCudaErrors(hipFree(d_normal));
	checkCudaErrors(hipFree(d_rank));
	return;
}

// int main(int argc, char **argv)
// {	
// 	//phaseran(result, SIGDIM, SIGSIZE);
// 	int viewers = 5;
// 	double *result = (double *)malloc(sizeof(double)*SIGSIZE*SIGDIM*viewers);	
// 	Timer phaseran_timer;
// 	phaseran_timer.Start();
// 	for(int i = 0; i <1 ; i++){

// 		for(int i = 0; i<viewers*SIGSIZE*SIGDIM;i++){			
// 			result[i] = (double) rand()/RAND_MAX;
// 		}
// 		double *d_result;
// 		hipMalloc(&d_result, sizeof(double)*viewers*SIGSIZE*SIGDIM);
// 		hipMemcpy(d_result, result, sizeof(double)*viewers*SIGSIZE*SIGDIM, hipMemcpyHostToDevice);
		
// 		double *db_result;
// 		hipMalloc(&db_result, sizeof(double)*viewers*SIGSIZE*SIGDIM);

// 		amplitudeAdjustedFourierTransform(db_result, d_result, viewers, SIGDIM, SIGSIZE);
// 		hipMemcpy(result, d_result, sizeof(double)*viewers*SIGSIZE*SIGDIM, hipMemcpyDeviceToHost);
		
// 		hipFree(d_result);
// 		hipFree(db_result);
		
// 	}
// 	phaseran_timer.Pause();
// 	printf_timer(phaseran_timer);
// 	free(result);
// 	return 0;
// }	
